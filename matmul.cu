#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <utility>
#include <vector>

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Too slow to actually run!)
//
// void matmul_cpu_naive(
//     int32_t size_i,
//     int32_t size_j,
//     int32_t size_k,
//     float const *a,
//     float const *b,
//     float *c) {
//     for (int32_t i = 0; i < size_i; ++i) {
//         for (int32_t j = 0; j < size_j; ++j) {
//             float sum = 0.0;
//             for (int32_t k = 0; k < size_k; ++k) {
//                 sum += a[i * size_k + k] * b[k * size_j + j];
//             }
//             c[i * size_j + j] = sum;
//         }
//     }
// }

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (With Reuse in L1/Shmem)

namespace matmul_l1 {

__device__ void load_tile(
    float const *src, uint32_t src_width, 
    float *dst, uint32_t dst_height, uint32_t dst_width
) {
    for (uint32_t idx = threadIdx.x; idx < dst_height * dst_width; idx += blockDim.x) {
        // Get index to copy
        uint32_t i = idx / dst_width;
        uint32_t j = idx % dst_width;
        // Copy mem over
        dst[i * dst_width + j] = src[i * src_width + j];
    }
    // Wait for all the memory to be copied
    __syncthreads();
}

__device__ void matmul_tile(
    uint32_t size_i, uint32_t size_j, uint32_t size_k, // Matrix dimensions
    float const *a, float const *b, float *c, // Matrices
    float *local_a, float *local_b, // Matrices in SRAM
    uint32_t n // Tile size
) {
    // Math: c_ik = a_i ⋅ b_k
    // Goal: c_ik in registers and a_i, b_k in L1 cache
    // Plan: Each thread works on one c_ik at a time

    // Each thread gets a c_ik
    uint32_t i = threadIdx.x / n;
    uint32_t k = threadIdx.x % n;

    // Keep c_ik in local register
    float local_c_ik = 0.0f;

    uint32_t a_height = n;
    uint32_t a_width = min(size_j, 390);
    uint32_t b_height = min(size_j, 390);
    uint32_t b_width = n;

    // Iterate over subtiles of a_i, b_k
    for (uint32_t idx = 0; idx < size_j; idx += a_width) {
        // Update a_width, b_height
        a_width = min(a_width, size_j - idx);
        b_height = min(b_height, size_j - idx);

        // Load subtiles
        load_tile(
            a, size_j,
            local_a, a_height, a_width
        );
        load_tile(
            b, size_k,
            local_b, b_height, b_width
        );

        // Iterate over a_i, b_k
        for (uint32_t j = 0; j < a_width; ++j) {
            local_c_ik += local_a[i * a_width + j] * local_b[j * b_width + k];
        }

        // Move buffers
        a += a_width;
        b += b_height * size_k;

        __syncthreads();
    }

    // Write back to main memory at the end
    c[i * size_k + k] = local_c_ik;
}

__global__ void matmul_l1(
    int32_t size_i, int32_t size_j, int32_t size_k,
    float const *a, float const *b, float *c
) {
    // c_ik tiles are 32x32
    constexpr uint32_t n = 32;

    // Grid dimensions
    uint32_t tiles_per_row = size_k / n;
    uint32_t tiles_per_col = size_j / n;

    // Setup the block's SRAM
    extern __shared__ float sram[];
    float *local_a = sram;
    float *local_b = sram + 32 * min(size_j, 390); // 25000/(2*32) = 390

    // Iterate over tiles
    for (uint32_t idx = blockIdx.x; idx < tiles_per_col * tiles_per_row; idx += gridDim.x) {
        // Tile indices
        // uint32_t tile_i = idx / tiles_per_row; // Row major
        // uint32_t tile_k = idx % tiles_per_row;
        uint32_t tile_k = idx / tiles_per_col; // Col major
        uint32_t tile_i = idx % tiles_per_col;

        // Move buffers
        float const *tile_a = a + tile_i * n * size_j;
        float const *tile_b = b + tile_k * n;
        float *tile_c = c + tile_i * n * size_j + tile_k * n;

        matmul_tile(
            size_i, size_j, size_k,
            tile_a, tile_b, tile_c,
            local_a, local_b,
            n
        );
    }
}

void launch_matmul_l1(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a,
    float const *b,
    float *c) {
    // Setup the block SRAM
    int shmem_size_bytes = 100 * 1013; // Max 100 KB per block
    CUDA_CHECK(hipFuncSetAttribute(
        reinterpret_cast<const void*>(matmul_l1),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        shmem_size_bytes
    ));
    matmul_l1<<<48, 32 * 32, shmem_size_bytes>>>(size_i, size_j, size_k, a, b, c);
}

// Part 2 lower bound: 17ms

}; // namespace matmul_l1

////////////////////////////////////////////////////////////////////////////////
// GPU Implementation (With Reuse in L1/Shmem and Registers)

namespace matmul_l1_reg {

__global__ void matmul_l1_reg(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a,
    float const *b,
    float *c) {
    /* TODO: your GPU code here */
}

void launch_matmul_l1_reg(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a,
    float const *b,
    float *c) {
    /* TODO: your CPU code here */
    // Lower bound: 5ms
}

}; // namespace matmul_l1_reg

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

std::vector<float> read_data(std::string const &path, int32_t size) {
    std::ifstream file(path, std::ios::binary);
    std::vector<float> data(size);
    file.read(reinterpret_cast<char *>(data.data()), data.size() * sizeof(float));
    if (file.fail()) {
        std::cerr << "Failed to read " << path << std::endl;
        std::abort();
    }
    return data;
}

template <typename F>
double benchmark_ms(double target_time_ms, int32_t num_iters_inner, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        for (int32_t i = 0; i < num_iters_inner; ++i) {
            f();
        }
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms / num_iters_inner);
    }
    return best_time_ms;
}

struct BenchmarkResult {
    char const *name;
    double elapsed_ms;
};

struct BenchmarkConfig {
    int32_t size_i;
    int32_t size_j;
    int32_t size_k;
    bool save_result;
};

template <typename Impl>
void run_tests_for_size(
    std::string const &test_data_dir,
    std::vector<BenchmarkResult> &saved_results,
    std::vector<BenchmarkConfig> const &configs) {
    for (auto config : configs) {
        auto size_i = config.size_i;
        auto size_j = config.size_j;
        auto size_k = config.size_k;

        auto path_prefix = test_data_dir + "/test_" + std::to_string(size_i) + "x" +
            std::to_string(size_j) + "x" + std::to_string(size_k);
        auto a = read_data(path_prefix + "_a.bin", size_i * size_k);
        auto b = read_data(path_prefix + "_b.bin", size_k * size_j);
        auto c = read_data(path_prefix + "_c.bin", size_i * size_j);

        float *a_gpu;
        float *b_gpu;
        float *c_gpu;
        CUDA_CHECK(hipMalloc(&a_gpu, size_i * size_k * sizeof(float)));
        CUDA_CHECK(hipMalloc(&b_gpu, size_k * size_j * sizeof(float)));
        CUDA_CHECK(hipMalloc(&c_gpu, size_i * size_j * sizeof(float)));

        CUDA_CHECK(hipMemcpy(
            a_gpu,
            a.data(),
            size_i * size_k * sizeof(float),
            hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(
            b_gpu,
            b.data(),
            size_k * size_j * sizeof(float),
            hipMemcpyHostToDevice));

        Impl::run(size_i, size_j, size_k, a_gpu, b_gpu, c_gpu);

        std::vector<float> c_out_host(size_i * size_j);
        CUDA_CHECK(hipMemcpy(
            c_out_host.data(),
            c_gpu,
            size_i * size_j * sizeof(float),
            hipMemcpyDeviceToHost));

        double mse = 0.0;
        double ref_mean_square = 0.0;
        for (int32_t i = 0; i < size_i; ++i) {
            for (int32_t j = 0; j < size_j; ++j) {
                float diff = c_out_host[i * size_j + j] - c[i * size_j + j];
                mse += diff * diff;
                ref_mean_square += c[i * size_j + j] * c[i * size_j + j];
            }
        }
        mse /= size_i * size_j;
        ref_mean_square /= size_i * size_j;
        float rmse = std::sqrt(mse);
        float rel_rmse = rmse / std::sqrt(ref_mean_square);

        printf("  size %4d * %4d * %4d:\n", size_i, size_j, size_k);
        printf("    correctness: %.02e relative RMSE\n", rel_rmse);

        if (rel_rmse > 1e-5) {
            printf("    skipping benchmark (incorrect)\n");
        } else {
            double elapsed_ms = benchmark_ms(1000.0, 4, [&]() {
                Impl::run(size_i, size_j, size_k, a_gpu, b_gpu, c_gpu);
            });

            printf("    run time: %6.02f ms\n", elapsed_ms);

            double tflop = 2.0 * size_i * size_k * size_j * 1e-12;
            printf("    throughput: %5.02f TFLOP/s\n", tflop / (elapsed_ms * 1e-3));

            if (config.save_result) {
                saved_results.push_back({Impl::name, elapsed_ms});
            }
        }

        printf("\n");
    }
}

template <typename Impl>
void run_all_tests(
    std::string const &test_data_dir,
    std::vector<BenchmarkResult> &saved_results) {
    printf("%s:\n\n", Impl::name);
    run_tests_for_size<Impl>(test_data_dir, saved_results, {{256, 256, 256, false}});
    run_tests_for_size<Impl>(test_data_dir, saved_results, {{3072, 3072, 3072, true}});
}

struct MatmulL1 {
    constexpr static char const *name = "matmul_l1";
    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c) {
        matmul_l1::launch_matmul_l1(size_i, size_j, size_k, a, b, c);
    }
};

struct MatmulL1Reg {
    constexpr static char const *name = "matmul_l1_reg";
    static void
    run(int32_t size_i,
        int32_t size_j,
        int32_t size_k,
        float const *a,
        float const *b,
        float *c) {
        matmul_l1_reg::launch_matmul_l1_reg(size_i, size_j, size_k, a, b, c);
    }
};

int main(int argc, char **argv) {
    std::string test_data_dir = ".";

    auto saved_results = std::vector<BenchmarkResult>();

    run_all_tests<MatmulL1>(test_data_dir, saved_results);
    run_all_tests<MatmulL1Reg>(test_data_dir, saved_results);

    if (saved_results.size() > 1) {
        printf("speedups on largest problem size:\n");
        for (int32_t j = 1; j < saved_results.size(); ++j) {
            printf("\n");
            for (int32_t i = j; i > 0;) {
                --i;
                auto const &first = saved_results.at(i);
                auto const &second = saved_results.at(j);
                printf(
                    "  speedup %s -> %s: %.02fx\n",
                    first.name,
                    second.name,
                    first.elapsed_ms / second.elapsed_ms);
            }
        }
    }

    return 0;
}
